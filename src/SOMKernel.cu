#include "hip/hip_runtime.h"
#ifndef _SOMKERNELS_
#define _SOMKERNELS_

#include "include/math/Functions.h"
#include "include/math/Random.h"
#include "include/gpgpu/Kernels.h"
#include <cfloat>

#include <cassert>
#include <cmath>


struct saxmy_functor {
	const float a;

	saxmy_functor(float _a) : a(_a) {}

	__host__ __device__
	float operator()(const float& x, const float& y) const { 
		return a * (x - y);
	}
};


// return the biggest of two tuples
struct bigger_tuple_functor {
    __device__ __host__
    thrust::tuple<float, unsigned int> operator() (	
    	const thrust::tuple<float, unsigned int> &a, 
		const thrust::tuple<float, unsigned int> &b ) 
    {
    	return (a >= b) ? a : b;
    }
};

// return the biggest of two tuples
struct smaller_tuple_functor {
    __device__ __host__
    thrust::tuple<float, unsigned int> operator() (	
    	const thrust::tuple<float, unsigned int> &a, 
		const thrust::tuple<float, unsigned int> &b ) 
    {
    	return (a <= b) ? a : b;
    }
};

float hostGetMax(const thrust::device_vector<float>& vec, unsigned int &ID) {
    // create implicit index sequence [0, 1, 2, ... ]
	thrust::counting_iterator<unsigned int> begin(0);
	thrust::counting_iterator<unsigned int> end(vec.size() );

    thrust::tuple<float, unsigned int> init(vec[0], 0);
    thrust::tuple<float, unsigned int> smallest;

    smallest = reduce( thrust::make_zip_iterator(make_tuple(vec.begin(), begin) ),
    				   thrust::make_zip_iterator(make_tuple(vec.end(), end) ),
                       init,
                       bigger_tuple_functor() );

    ID = thrust::get<1>(smallest);
    return vec[ID];
}

float hostGetMin(const thrust::device_vector<float>& vec, unsigned int &ID) {
    // create implicit index sequence [0, 1, 2, ... ]
	thrust::counting_iterator<unsigned int> begin(0);
	thrust::counting_iterator<unsigned int> end(vec.size() );
	
	thrust::tuple<float, unsigned int> init(vec[0], 0);
	thrust::tuple<float, unsigned int> smallest;
	
	smallest = reduce( thrust::make_zip_iterator(make_tuple(vec.begin(), begin) ),
					   thrust::make_zip_iterator(make_tuple(vec.end(), end) ),
					   init,
					   smaller_tuple_functor() );

	ID = thrust::get<1>(smallest);
    return vec[ID];
}
//////////////////////////////////////////////////////////////////////////////////////////////

struct minus_pow_functor {
    const float fVal;
    minus_pow_functor(float val) : fVal(val) {}

    __host__ __device__
	float operator()(const float& val) const { 
		return pow(fVal-val, 2);
	}
};

struct sqrt_functor {
    __host__ __device__
	float operator()(const float& val) const { 
		return sqrt(val);
	}
};
//////////////////////////////////////////////////////////////////////////////////////////////

struct gaussian_bell_functor {
	float fSigmaT;
	gaussian_bell_functor(const float &sigmaT) : fSigmaT(sigmaT)	{}

    __host__ __device__
	float operator()(const float& dist) const {
    	return ANN::fcn_gaussian_bell(dist, fSigmaT);
	}
};

struct hebbian_functor {
	float fLearningRate;
	float fInput;

	hebbian_functor(const float &learning_rate, const float &input) :
		fLearningRate(learning_rate), fInput(input) {}

    __host__ __device__
	float operator()(const float& fWeight, const float& fInfluence) const {
    	return fWeight + (fInfluence*fLearningRate*(fInput-fWeight) );
	}
};

/*
 * Layout of SOMEdgeMatrix:
 * 			COL1	COL2	COL3	COL(n+1)
 * ROW1		toNeur1	toNeur1	toNeur1	..
 * ROW2		toNeur2	toNeur2	toNeur2	..
 * ROW3		toNeur3	toNeur3	toNeur3	..
 * ROW(n+1)	..		..		..
 */
BMUExport
hostSOMFindBMNeuronID(std::vector<SplittedNetExport> &SExp,
		const thrust::device_vector<float> &InputVector,
		const float &fConscienceRate)
{
	BMUExport retBMU;
	float fLastBMU = FLT_MAX;

	#pragma omp parallel for
	for(int iDev = 0; iDev < static_cast<int>(SExp.size() ); iDev++) {
		if(hipSetDevice(iDev) != hipSuccess) {
			std::cout<<"hostSOMTraining(): Setting new cuda-capable device failed."<<std::endl;
			continue;
		} else {
			unsigned int BMUID = 0;

			unsigned int iWidth 	= SExp.at(iDev).f2dEdges.getW();
			unsigned int iHeight 	= SExp.at(iDev).f2dEdges.getH();

			assert(iWidth > 0);
			assert(iHeight > 0);

			thrust::device_vector<float> dvRes(iWidth, 0.f);
			thrust::device_vector<float> dvTmp(iWidth, 0.f);// temporary

			for(unsigned int y = 0; y < iHeight; y++) {
				thrust::transform(
					SExp.at(iDev).f2dEdges.getRowBegin(y),	// input
					SExp.at(iDev).f2dEdges.getRowEnd(y), 	// input
					dvTmp.begin(), 							// result
					minus_pow_functor(InputVector[y]) ); 	// functor

				thrust::transform(
					dvRes.begin(), 						// input
					dvRes.end(), 						// input
					dvTmp.begin(),						// input
					dvRes.begin(), 						// result
					thrust::plus<float>() );			// functor
			}
			dvTmp = dvRes;

			// implementation of conscience mechanism
			if(fConscienceRate > 0.f) {
				thrust::device_vector<float> dvConscience(iWidth, 1.f / (float)iWidth);

				thrust::transform(
					dvConscience.begin(),
					dvConscience.end(),
					SExp.at(iDev).dvConscience.begin(),
					dvConscience.begin(),
					thrust::minus<float>() );

				thrust::transform(
					dvRes.begin(),
					dvRes.end(),
					dvConscience.begin(),
					dvRes.begin(),
					thrust::minus<float>() );
			}

			thrust::transform(
				dvTmp.begin(),
				dvTmp.end(),
				SExp.at(iDev).dvConscience.begin(),
				SExp.at(iDev).dvConscience.begin(),
				saxmy_functor(fConscienceRate) );

			hostGetMin(dvRes, BMUID);

			// Check partial results for global BMU in all devices
			if(fLastBMU > dvRes[BMUID]) {
				fLastBMU = dvRes[BMUID];

				thrust::host_vector<float> vPos = SExp.at(iDev).f2dPositions.getCol(BMUID);
				retBMU = BMUExport(BMUID, iDev, vPos);
			}
		}
	}
	
	return retBMU;
}

/*
 * Layout of SOMPositionMatrix:
 * 			COL1	COL2	COL3	COL(n+1)
 * ROW1		Xpos	Xpos	Xpos	..
 * ROW2		Ypos	Ypos	Ypos	..
 * ROW3		Zpos	Zpos	Zpos	..
 * ROW(n+1)	..		..		..		..
 */
void hostSOMPropagateBW( std::vector<SplittedNetExport> &SExp,
		const thrust::device_vector<float> &dvInputVector,
		const BMUExport &BMU,
		const float &fSigmaT,
		const float &fLearningRate
		)
{
	#pragma omp parallel for
	for(int iDev = 0; iDev < static_cast<int>(SExp.size() ); iDev++) {
		if(hipSetDevice(iDev) != hipSuccess) {
			std::cout<<"hostSOMTraining(): Setting new cuda-capable device failed."<<std::endl;
			continue;
		} else {
			unsigned int iWidth 	= SExp.at(iDev).f2dPositions.getW();
			unsigned int iHeight 	= SExp.at(iDev).f2dPositions.getH();

			thrust::device_vector<float> dvBMUPos = BMU.dvBMUPos;
			thrust::device_vector<float> dvTmp(iWidth, 0.f); // temporary
			thrust::device_vector<float> dvInfluence(iWidth, 0.f);
			thrust::device_vector<float> dvDist(iWidth, 0.f);

			// 1. Calc distances for all neurons to BMNeuron
			// Distance = sqrt(pow(x,2)+pow(y,2)+pow(z,2)+pow(n+1,2) );
			for(unsigned int y = 0; y < iHeight; y++) { 	// for each coordinate position of the neuron
				thrust::transform(
					SExp.at(iDev).f2dPositions.getRowBegin(y),	// input
					SExp.at(iDev).f2dPositions.getRowEnd(y), 	// input
					dvTmp.begin(), 						// result
					minus_pow_functor(dvBMUPos[y]) ); 	// functor

				thrust::transform(
					dvDist.begin(), 					// input
					dvDist.end(), 						// input
					dvTmp.begin(),						// input
					dvDist.begin(), 					// result
					thrust::plus<float>() );			// functor
			}
			thrust::transform(
				dvDist.begin(),							// input
				dvDist.end(), 							// input
				dvDist.begin(), 						// result
				sqrt_functor() );						// functor

			// 2. Calculate the influence for each neuron
			thrust::transform(
				dvDist.begin(),							// input
				dvDist.end(), 							// input
				dvInfluence.begin(), 					// result
				gaussian_bell_functor(fSigmaT) );		// functor

			// 3. Only handle neurons in radius:
			// 3a. Make stencil
			dvTmp.assign(iWidth, fSigmaT);
			thrust::transform(
				dvDist.begin(), 						// input 1
				dvDist.end(),							// input 1
				dvTmp.begin(),							// input 1
				dvTmp.begin(), 							// result
				thrust::less_equal<float>() 			// functor
			);

			// 3b. Use stencil to modify only neurons inside the radius
			// Save result in the ANN::Matrix
			iWidth 	= SExp.at(iDev).f2dEdges.getW();
			iHeight = SExp.at(iDev).f2dEdges.getH();

			for(unsigned int y = 0; y < iHeight; y++) {			// for each edge of the neuron
				thrust::transform_if(
					SExp.at(iDev).f2dEdges.getRowBegin(y),		// input 1
					SExp.at(iDev).f2dEdges.getRowEnd(y), 		// input 1
					dvInfluence.begin(),						// input 2
					dvTmp.begin(),								// stencil
					SExp.at(iDev).f2dEdges.getRowBegin(y), 		// result
					hebbian_functor(fLearningRate, dvInputVector[y]), // functor
					thrust::identity<int>() ); 					// predicate
			}
		}
	}
}

void hostSOMTraining( std::vector<SplittedNetExport> &SExp,
		const ANN::TrainingSet &InputSet,
		const unsigned int &iCycles,
		const float &fSigma0, 
		const float &fLearningRate0,
		const float &fConscienceRate,
		float (*pfnDecay)(const float &, const float &, const float &) )
{
	float fLambda 	= iCycles / log(fSigma0);

	int iMin 		= 0;
	int iMax 		= InputSet.GetNrElements()-1;
	unsigned int iProgCount = 1;

	// use 8 proximal neurons as standard
	float fSigmaT = sqrt(2.f);

	for(unsigned int i = 0; i < iCycles; i++) {
		if(iCycles >= 10) {
			if(((i+1) / (iCycles/10)) == iProgCount && (i+1) % (iCycles/10) == 0) {
				std::cout<<"Current training progress calculated by the GPU is: "<<iProgCount*10.f<<"%/Step="<<i+1<<std::endl;
				iProgCount++;
			}
		}
		else {
			std::cout<<"Current training progress calculated by the CPU is: "<<(float)(i+1.f)/(float)iCycles*100.f<<"%/Step="<<i+1<<std::endl;
		}
		// Set input
		std::vector<float> vCurInput = InputSet.GetInput(ANN::RandInt(iMin, iMax) );
		thrust::device_vector<float> dvInputVector(vCurInput.size() );
		thrust::copy(vCurInput.begin(), vCurInput.end(), dvInputVector.begin() );

		// Find BMNeuron
		BMUExport BMUExp;
		BMUExp = hostSOMFindBMNeuronID(SExp, dvInputVector, fConscienceRate);

		// Calc m_fSigmaT if conscience is _not_ used
		if(fConscienceRate <= 0.f)
			fSigmaT = pfnDecay(fSigma0, i, fLambda);
		float fLearningRate = pfnDecay(fLearningRate0, i, iCycles);

		// Propagate BW
		hostSOMPropagateBW( SExp,
				dvInputVector,		// const
				BMUExp,				// const
				fSigmaT,			// const
				fLearningRate ); 	// const
	}
}

#endif
