#include "hip/hip_runtime.h"
#ifndef _BPKERNELS_
#define _BPKERNELS_

#include "include/math/Functions.h"
#include "include/gpgpu/Kernels.h"
#include "include/math/Functions.h"


// Y <- A * X + Y
struct saxpy_functor {
    const float a;

    saxpy_functor(float _a) : a(_a) {}

    __host__ __device__
	float operator()(const float& x, const float& y) const {
		return a * x + y;
	}
};

// Y <- A * X * Y
struct sax_functor {
    const float a;

    sax_functor(float _a) : a(_a) {}

    __host__ __device__
	float operator()(const float& x) const {
		return a * x;
	}
};
///////////////////////////////////////////////////////////////////////

inline void
SwitchTransfFunc(	std::vector<thrust::device_vector<float> > &vNeuronValues,
					thrust::device_vector<float> &dvLayer,
					thrust::device_vector<float> &dvBias,
					thrust::device_vector<float> &dvInput,
					const ANN::TransfFunction &function)
{
	// Run values through transfer function
	if (strcmp(function.name, "tanh") == 0) {
	    thrust::transform(
	    		dvLayer.begin(),
	    		dvLayer.end(),
	    		dvBias.begin(),
	    		dvLayer.begin(),
	    		ANN::tanTransferFcn() );
		// Now the input of the next layer will be the the previous one
	    dvInput = dvLayer;
		vNeuronValues.push_back(dvLayer);
		return;
	}
	if (strcmp(function.name, "log") == 0) {
	    thrust::transform(
	    		dvLayer.begin(),
	    		dvLayer.end(),
	    		dvBias.begin(),
	    		dvLayer.begin(),
	    		ANN::logTransferFcn() );
		// Now the input of the next layer will be the the previous one
	    dvInput = dvLayer;
		vNeuronValues.push_back(dvLayer);
		return;
	}
	if (strcmp(function.name, "binary") == 0) {
	    thrust::transform(
	    		dvLayer.begin(),
	    		dvLayer.end(),
	    		dvBias.begin(),
	    		dvLayer.begin(),
	    		ANN::binTransferFcn() );
		// Now the input of the next layer will be the the previous one
	    dvInput = dvLayer;
		vNeuronValues.push_back(dvLayer);
		return;
	}
	if (strcmp(function.name, "linear") == 0) {
	    thrust::transform(
	    		dvLayer.begin(),
	    		dvLayer.end(),
	    		dvBias.begin(),
	    		dvLayer.begin(),
	    		ANN::linTransferFcn() );
		// Now the input of the next layer will be the the previous one
	    dvInput = dvLayer;
		vNeuronValues.push_back(dvLayer);
		return;
	}
}

inline void
SwitchDevTransfFunc(thrust::device_vector<float> &dvNeurons,
					const std::vector<thrust::device_vector<float> > &vNeuronValues,
					const ANN::TransfFunction &function,
					const int &i)
{
	if (strcmp(function.name, "tanh") == 0) {
	    thrust::transform(
				vNeuronValues.at(i).begin(),
				vNeuronValues.at(i).end(),
				dvNeurons.begin(),
	    		ANN::devTanTransferFcn() );
	    return;
	}
	if (strcmp(function.name, "log") == 0) {
	    thrust::transform(
				vNeuronValues.at(i).begin(),
				vNeuronValues.at(i).end(),
				dvNeurons.begin(),
	    		ANN::devLogTransferFcn() );
	    return;
	}
	if (strcmp(function.name, "binary") == 0) {
				thrust::transform(
				vNeuronValues.at(i).begin(),
				vNeuronValues.at(i).end(),
				dvNeurons.begin(),
	    		ANN::devBinTransferFcn() );
	    return;
	}
	if (strcmp(function.name, "linear") == 0) {
	    thrust::transform(
				vNeuronValues.at(i).begin(),
				vNeuronValues.at(i).end(),
				dvNeurons.begin(),
	    		ANN::devLinTransferFcn() );
	    return;
	}
}
///////////////////////////////////////////////////////////////////////

std::vector<float>
hostBPCalcDelta(	const thrust::device_vector<float> &dvNeurOut,	// from forward run
					const std::vector<float> &vTrainOut ) 			// from training set
{
	thrust::device_vector<float> dvTrainOut (vTrainOut.begin(), vTrainOut.end() );
	thrust::device_vector<float> dvDelta	(vTrainOut.size(), 0.f);
    std::vector<float> vRes(vTrainOut.size() );

	// Calc error deltas of output layer
    thrust::transform(
    		dvTrainOut.begin(),
    		dvTrainOut.end(),
    		dvNeurOut.begin(),
    		dvDelta.begin(),
    		thrust::minus<float>() );

    thrust::copy(dvDelta.begin(), dvDelta.end(), vRes.begin());
    return vRes;
}
///////////////////////////////////////////////////////////////////////

std::vector<thrust::device_vector<float> >
hostBPPropagateFW(	const std::vector<ANN::Matrix> &vEdgeMatrices,
					const std::vector<ANN::Matrix> &vBiasEdgeMatrices,
					const std::vector<float> &vInput,
					const ANN::TransfFunction &function)
{
	std::vector<thrust::device_vector<float> > vNeuronValues(1, vInput);

	// Copy Input from vInput in device vector: vOutput
	thrust::device_vector<float> dvInput(vInput.begin(), vInput.end() ); 	// input
	thrust::device_vector<float> dvLayer;
	thrust::device_vector<float> dvBias;
	
	unsigned int iWidth 	= 0;
	unsigned int iHeight 	= 0;

	for(unsigned int i = 0; i < vEdgeMatrices.size(); i++) {	
		iWidth 		= vEdgeMatrices.at(i).getW();
		iHeight 	= vEdgeMatrices.at(i).getH();
		
		// Alloc memory
		dvLayer 	= thrust::device_vector<float>(iWidth, 0.f);
		dvBias 		= thrust::device_vector<float>(iWidth, 0.f);
	
		if(vBiasEdgeMatrices.at(i).getW() > 0) {
			dvLayer = thrust::device_vector<float>(vBiasEdgeMatrices.at(i).getRowBegin(0), vBiasEdgeMatrices.at(i).getRowEnd(0));
			dvBias 	= thrust::device_vector<float>(vBiasEdgeMatrices.at(i).getRowBegin(0), vBiasEdgeMatrices.at(i).getRowEnd(0));

			// initial bias term
			thrust::transform(dvBias.begin(),
				dvBias.end(),
				dvLayer.begin(),
				thrust::negate<float>() );

			// bias weights
		    thrust::transform( dvBias.begin(),
		    		dvBias.end(),
		    		dvLayer.begin(),
		    		dvLayer.begin(),
		    		saxpy_functor(1) );
		}

		// Calculate the result of the current layer
		for(unsigned int y = 0; y < iHeight; y++) {
		    // Y <- A * X + Y
		    thrust::transform( vEdgeMatrices.at(i).getRowBegin(y),
		    		vEdgeMatrices.at(i).getRowEnd(y),
		    		dvLayer.begin(),
		    		dvLayer.begin(),
		    		saxpy_functor(dvInput[y]) );
		}

		SwitchTransfFunc( vNeuronValues, dvLayer, dvBias, dvInput, function );
	}
	return vNeuronValues;
}
///////////////////////////////////////////////////////////////////////

inline void
AdadtEdges(	std::vector<ANN::Matrix> &vEdgeMatricesI,
			std::vector<thrust::device_vector<float> > &vErrors,
			std::vector<ANN::Matrix> &vMomentums,
			const std::vector<thrust::device_vector<float> > &vNeuronValues,
			const float &fLearningRate,
			const float &fWeightDecay,
			const float &fMomentum,
			const unsigned int iWidth, const unsigned int iHeight, const unsigned int i)
{
	/*
	 * Quick standard implementation
	 */
	if(fWeightDecay == 0.f && fMomentum == 0.f) {
		for(unsigned int y = 0; y < iHeight; y++) {
			thrust::transform( vErrors.at(i+1).begin(),
				vErrors.at(i+1).end(),
				vEdgeMatricesI.at(i).getRowBegin(y),
				vEdgeMatricesI.at(i).getRowBegin(y),
				saxpy_functor(fLearningRate*vNeuronValues.at(i)[y]) );
		}
		return;
	}

	/*
	 * Slower but more complex one
	 */
	thrust::device_vector<float> dvMomentums(iWidth, 0.f);
	ANN::Matrix matMomentums(iWidth, iHeight, 0);
	if(!vMomentums.size()) {
		vMomentums = std::vector<ANN::Matrix>(iHeight);
	}

	for(unsigned int y = 0; y < iHeight; y++) {
		// standard term
		thrust::transform( vErrors.at(i+1).begin(),
			vErrors.at(i+1).end(),
			dvMomentums.begin(),
			sax_functor(fLearningRate*vNeuronValues.at(i)[y]) );
		// weight decay
		if(fWeightDecay > 0.f) {
			thrust::transform( vEdgeMatricesI.at(i).getRowBegin(y),
				vEdgeMatricesI.at(i).getRowEnd(y),
				dvMomentums.begin(),
				dvMomentums.begin(),
				saxpy_functor(-fWeightDecay) );
		}
		// momentum term
		if(vMomentums.at(y).size() && fMomentum > 0.f) {
			thrust::transform( vMomentums.at(i).getRowBegin(y),
				vMomentums.at(i).getRowEnd(y),
				dvMomentums.begin(),
				dvMomentums.begin(),
				saxpy_functor(fMomentum) );

			thrust::copy(dvMomentums.begin(), dvMomentums.end(), matMomentums.getRowBegin(y) );
		}
		// .. belongs to standard term and updates weights
		thrust::transform( dvMomentums.begin(),
			dvMomentums.end(),
			vEdgeMatricesI.at(i).getRowBegin(y),
			vEdgeMatricesI.at(i).getRowBegin(y),
			thrust::plus<float>() );
	}
	// Safe momentums for the next run
	if(fMomentum > 0.f) {
		vMomentums[i] = matMomentums;
	}
}

void
hostBPPropagateBW(	std::vector<ANN::Matrix> &vEdgeMatricesI,
					std::vector<ANN::Matrix> &vMomentums,
					std::vector<thrust::device_vector<float> > &vErrors,
					const std::vector<thrust::device_vector<float> > &vNeuronValues,
					const float &fLearningRate,
					const float &fWeightDecay,
					const float &fMomentum,
					const ANN::TransfFunction &function )
{
	// All layers except output!
	for(int i = vEdgeMatricesI.size()-1; i >= 0; i--) {
		unsigned int iWidth 	= vEdgeMatricesI.at(i).getW();
		unsigned int iHeight 	= vEdgeMatricesI.at(i).getH();

		if(iWidth == 0 || iHeight == 0) {
			continue;
		}

		// errors of this layer
		assert(vErrors.at(i).size() == vNeuronValues.at(i).size());
		thrust::device_vector<float> dvErrors(vErrors.at(i).size(), 0);
		thrust::device_vector<float> dvNeurons(vNeuronValues.at(i).size(), 0);
		thrust::device_vector<float> dvEdges(iWidth, 0);

		// Calculate the result of the current layer
		for(unsigned int y = 0; y < iHeight; y++) {
			thrust::transform( vEdgeMatricesI.at(i).getRowBegin(y),
				vEdgeMatricesI.at(i).getRowEnd(y),
				vErrors.at(i+1).begin(),
				dvEdges.begin(),
				thrust::multiplies<float>() );

			dvErrors[y] = thrust::reduce(dvEdges.begin(), dvEdges.end(), (float) 0, thrust::plus<float>());
		}

		thrust::transform( vNeuronValues.at(i).begin(),
			vNeuronValues.at(i).end(),
			dvNeurons.begin(),
			ANN::devLogTransferFcn() );

		SwitchDevTransfFunc( dvNeurons, vNeuronValues, function, i );

		thrust::transform( dvNeurons.begin(),
			dvNeurons.end(),
			dvErrors.begin(),
			vErrors.at(i).begin(),
			thrust::multiplies<float>() );
	}

	// All layers except output ..
	for(int i = vEdgeMatricesI.size()-1; i >= 0 && vNeuronValues.size() > 0; i--) {
		unsigned int iWidth 	= vEdgeMatricesI.at(i).getW();
		unsigned int iHeight 	= vEdgeMatricesI.at(i).getH();

		AdadtEdges( vEdgeMatricesI, vErrors, vMomentums, vNeuronValues,
					fLearningRate, fWeightDecay, fMomentum,
					iWidth, iHeight, i );
	}
}

#endif
